#include <hip/hip_runtime.h>

__global__
void saxpy_kernel(int n, float a, float *x, float *y)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if ( i < n )
    y[i] += a * x[i];
}
extern "C" void saxpy(int n ,float a, float *x, float *y, hipStream_t stream)
{
  dim3 griddim, blockdim;

  blockdim = dim3(128,1,1);
  griddim = dim3(n/blockdim.x,1,1);

  saxpy_kernel<<<griddim,blockdim,0,stream>>>(n,a,x,y);
}
